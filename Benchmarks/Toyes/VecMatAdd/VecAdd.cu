/**
 * This is a toy program from CUDA tutorial, it is to compute the addition on
 * two square matrixes, Here I use it to study some basic built-in data
 * structures provided by CUDA
 */


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define N 512
#define BLK 32

__global__ void VecAdd(float *A, float *B, float *C) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  C[i] = A[i] + B[i];
}

int main(int argc, char **argv) {
  // A and B are input data, C is to store output data
  float A[N], B[N], C[N];

  size_t size = N * sizeof(float);

  printf("size: %ld\n", size);

  // initialize the input data
  for (int i = 0; i < N; i++) {
    A[i] = sin(i) * sin(i);
    B[i] = cos(i) * cos(i);
  }

  // correspoinding memory for data in device
  float *d_A, *d_B, *d_C;
  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);
  hipMalloc(&d_C, size);

  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  dim3 ThreadsPerBlock(BLK);
  dim3 BlocksPerGrid(N / BLK);
  VecAdd<<<BlocksPerGrid, ThreadsPerBlock>>>(d_A, d_B, d_C);

  hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

  for (int i = 0; i < BLK; i++) {
    printf("%.2f ", C[i]);
  }
  printf("\n");

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  hipMalloc(&d_A, N * sizeof(float));
  hipMalloc(&d_B, N * sizeof(float));
  hipMalloc(&d_C, N * sizeof(float));

  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  return 0;
}