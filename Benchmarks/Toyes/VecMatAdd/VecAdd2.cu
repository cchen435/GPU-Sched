/**
 * This is a toy program from CUDA tutorial, it is to compute the addition on
 * two square matrixes, Here I use it to study some basic built-in data
 * structures provided by CUDA
 */


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

#define N 4096
#define BLK 32

__global__ void VecAdd(double *A, double *B, double *C) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  C[i] = A[i] + B[i];
}

int main(int argc, char **argv) {
  // A and B are input data, C is to store output data
  double *A, *B, *C, sum = 0;
  dim3 ThreadsPerBlock(BLK);
  dim3 BlocksPerGrid(N/BLK);
  size_t size = N * sizeof(double);

  // initialize the input data
  for (int i = 0; i < N; i++) {
    A[i] = sin(i) * sin(i);
    B[i] = cos(i) * cos(i);
  }

  // correspoinding memory for data in device
  double *d_src[2], *d_res;
  hipMalloc(&d_src[0], size);
  hipMalloc(&d_src[1], size);
  hipMalloc(&d_res, size);

  printf("Init: %p, %p, %p\n", d_src[0], d_src[1], d_res);

  hipMemcpy(d_src[0], A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_src[1], B, size, hipMemcpyHostToDevice);

  VecAdd<<<BlocksPerGrid, ThreadsPerBlock>>>(d_src[0], d_src[1], d_res);
  printf("Maturized: %p, %p, %p\n", d_src[0], d_src[1], d_res);

  // cudaMemcpy(d_src[1], d_res, size, cudaMemcpyDeviceToDevice);
  // VecAdd<<<BlocksPerGrid, ThreadsPerBlock>>>(d_src[0], d_src[1], d_res);

  hipMemcpy(C, d_res, size, hipMemcpyDeviceToHost);

  for (int i = 0; i < size; i++) sum+= C[i];
  printf("sum = %f\n", sum);


  hipFree(d_src[0]);
  hipFree(d_src[1]);
  hipFree(d_res);
  return 0;
}
